#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// n - k - nb - bs - b - shared_size
// V0 || V1 || V2 || V3
// 1:   8-2         ||  8-2-16-4            ||  8-2-4-4-2               ||  8-2-4-4-2-36
// 2:   5000-4      ||  4096-4-4096-4096    ||  4096-4-4096-64-8        ||  4096-4-4096-64-32-66564 // // 4096-4-4096-64-8-4356
// 3:   10000-10    ||  10000-10-50000-2000 ||  10000-10-5000-200-10    ||

#define n 4096
#define k 100

// v1
#define nb 4096
#define bs 4096

// v2
#define nb_v2 4096
#define bs_v2 64
#define b_v2 8

// v3
#define nb_v3 4096
#define bs_v3 64
#define b_v3 8
// type shared_size (sqrt((n*n)/nb_v3)+2)^2
#define shared_size 4356

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}
// Timer
struct timespec timerStart(struct timespec start)
{
    clock_gettime(CLOCK_MONOTONIC, &start);
    return start;
}
struct timespec timerStop(struct timespec stop)
{
    clock_gettime(CLOCK_MONOTONIC, &stop);
    return stop;
}
double timeDif(struct timespec start_, struct timespec stop_)
{
    double time_dif;
    time_dif = (stop_.tv_sec - start_.tv_sec) * 1e9;
    time_dif = (time_dif + (stop_.tv_nsec - start_.tv_nsec)) * 1e-9;
    return time_dif;
}

__global__ void v1_kernel(int* ising_sign_d_v1, int* ising_out_v1);

__global__ void v2_kernel(int* ising_sign_d_v2, int* ising_out_v2);

__global__ void v3_kernel(int* ising_sign_d_v3, int* ising_out_v3, int block_elems, int side_blocks, int side_block_elems);

int main(int argc, char* argv[])
{
    printf("implematation parameteres for n: %d k: %d\n\n", n, k);
    printf("V1| nb_v1: %d bs_v1: %d\n\n", nb, bs);
    printf("V2|| nb_v2: %d bs_v2: %d b_v2: %d\n\n", nb_v2, bs_v2, b_v2);
    printf("V3|| nb_v3: %d bs_v3: %d b_v3: %d\n\n", nb_v3, bs_v3, b_v3);

    int* sign;
    time_t t;

    /* Intializes random number generator */
    srand((unsigned)time(&t));

    int sign_size = (n + 2) * (n + 2);
    int ising_sign_size = n * n;
    // Malloc 1D Arrays

    sign = (int*)malloc(sign_size * sizeof(int));

    // Could use module but is better for CPI to surround the array with 1 line of values

    // Initialize 1D array

    for (int i = 0; i < sign_size; i++) {
        sign[i] = 1 - (2 * (rand() % 2));
    }

    struct timespec start_v0 = { 0 }, stop_v0 = { 0 };
    struct timespec start_v1 = { 0 }, stop_v1 = { 0 };
    struct timespec start_v2 = { 0 }, stop_v2 = { 0 };
    struct timespec start_v3 = { 0 }, stop_v3 = { 0 };

    printf("\nv0 starts\n\n");

    /*------------------------V0------------------------*/

    int* sign_v0;
    sign_v0 = (int*)malloc(sizeof(int) * sign_size);

    int* ising_sign_v0;
    ising_sign_v0 = (int*)malloc(sizeof(int) * ising_sign_size);

    for (int i = 0; i < sign_size; i++) {
        sign_v0[i] = sign[i];
    }

    start_v0 = timerStart(start_v0);
    for (int k_count = 0; k_count < k; k_count++) {
        // 1st column
        sign_v0[0] = 0;
        sign_v0[(n + 2) * (n + 1)] = 0;
        for (int l = 0, i = n + 2, j = n + 2 + n; l < n; l++) {
            sign_v0[i] = sign_v0[j];
            i += n + 2;
            j += n + 2;
        }

        // 1st row
        for (int l = 0, i = 1, j = n * (n + 2) + 1; l < n; l++) {
            sign_v0[i] = sign_v0[j];
            i++;
            j++;
        }

        // 2nd column
        sign_v0[n + 1] = 0;
        sign_v0[(n + 2) * (n + 2) - 1] = 0;
        for (int l = 0, i = n + 2 + n + 1, j = n + 2 + 1; l < n; l++) {
            sign_v0[i] = sign_v0[j];
            i += n + 2;
            j += n + 2;
        }

        // 2nd row
        for (int l = 0, i = (n + 2) * (n + 1) + 1, j = (n + 2) + 1; l < n; l++) {
            sign_v0[i] = sign_v0[j];
            i++;
            j++;
        }
        // print sign_v0 array
        // for (int i = 0; i < sign_size; i++) {
        //     printf("%d\t", sign_v0[i]);
        //     if ((i + 1) % (n + 2) == 0) {
        //         printf("\n");
        //     }
        // }
        // printf("\n\n");

        for (int i = 0, j = n + 3; i < ising_sign_size;) {
            // printf("\t");
            for (int t_i = 0; t_i < n; t_i++, i++, j++) {
                ising_sign_v0[i] = sign_v0[j] + sign_v0[j - 1] + sign_v0[j + 1] + sign_v0[j - n - 2] + sign_v0[j + n + 2];
                ising_sign_v0[i] /= abs(ising_sign_v0[i]);
                // printf("%d\t", ising_sign_v0[i]);
            }
            j += 2;
            // printf("\n");
        }
        for (int i = 0, j = n + 3; i < ising_sign_size;) {
            for (int t_i = 0; t_i < n; t_i++, i++, j++) {
                sign_v0[j] = ising_sign_v0[i];
            }
            j += 2;
        }
        // printf("\n");
    }
    stop_v0 = timerStop(stop_v0);
    // free memory
    free(sign_v0);
    free(ising_sign_v0);

    printf("Time for V0: %lf seconds\n", timeDif(start_v0, stop_v0));
    printf("\nv1 starts\n\n");

    /*------------------------V1------------------------*/

    int* sign_v1;
    int* ising_out_h_v1;

    // CUDA lines
    int* ising_sign_d_v1;
    int* ising_out_v1;

    sign_v1 = (int*)malloc(sign_size * sizeof(int));

    for (int i = 0; i < sign_size; i++) {
        sign_v1[i] = sign[i];
    }

    start_v1 = timerStart(start_v1);
    for (int k_count = 0; k_count < k; k_count++) {
        // 1st column
        sign_v1[0] = 0;
        sign_v1[(n + 2) * (n + 1)] = 0;
        for (int l = 0, i = n + 2, j = n + 2 + n; l < n; l++) {
            sign_v1[i] = sign_v1[j];
            i += n + 2;
            j += n + 2;
        }

        // 1st row
        for (int l = 0, i = 1, j = n * (n + 2) + 1; l < n; l++) {
            sign_v1[i] = sign_v1[j];
            i++;
            j++;
        }

        // 2nd column
        sign_v1[n + 1] = 0;
        sign_v1[(n + 2) * (n + 2) - 1] = 0;
        for (int l = 0, i = n + 2 + n + 1, j = n + 2 + 1; l < n; l++) {
            sign_v1[i] = sign_v1[j];
            i += n + 2;
            j += n + 2;
        }

        // 2nd row
        for (int l = 0, i = (n + 2) * (n + 1) + 1, j = (n + 2) + 1; l < n; l++) {
            sign_v1[i] = sign_v1[j];
            i++;
            j++;
        }

        // print sign_v1 array
        // for (int i = 0; i < sign_size; i++) {
        //     printf("%d\t", sign_v1[i]);
        //     if ((i + 1) % (n + 2) == 0) {
        //         printf("\n");
        //     }
        // }
        // printf("\n\n");

        // CUDA

        gpuErrchk(hipMalloc((void**)&ising_sign_d_v1, (n + 2) * (n + 2) * sizeof(int)));
        gpuErrchk(hipMemcpy(ising_sign_d_v1, sign_v1, (n + 2) * (n + 2) * sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMalloc((void**)&ising_out_v1, n * n * sizeof(int)));

        v1_kernel<<<nb, bs>>>(ising_sign_d_v1, ising_out_v1);
        ising_out_h_v1 = (int*)malloc(ising_sign_size * sizeof(int));
        gpuErrchk(hipMemcpy(ising_out_h_v1, ising_out_v1, ising_sign_size * sizeof(int), hipMemcpyDeviceToHost));

        // Print ising kernel output

        //  printf("\t");
        // for (int i = 0; i < (nb * bs); i++) {
        //     printf("%d\t", ising_out_h_v1[i]);
        //     if ((i % n) == n - 1) {
        //         printf("\n\t");
        //     }
        // }
        // printf("\n\n");
        for (int i = 0, j = n + 3; i < ising_sign_size;) {
            for (int t_i = 0; t_i < n; t_i++, i++, j++) {
                sign_v1[j] = ising_out_h_v1[i];
            }
            j += 2;
        }

        // Free cuda memory
        hipFree(ising_sign_d_v1);
        hipFree(ising_out_v1);
    }
    stop_v1 = timerStop(stop_v1);

    free(sign_v1);

    printf("Time for V1: %lf seconds\n", timeDif(start_v1, stop_v1));

    printf("\nv2 starts\n\n");

    /*------------------------V2------------------------*/

    int* sign_v2;
    int* ising_out_h_v2;

    // CUDA lines
    int* ising_sign_d_v2;
    int* ising_out_v2;

    sign_v2 = (int*)malloc(sign_size * sizeof(int));

    for (int i = 0; i < sign_size; i++) {
        sign_v2[i] = sign[i];
    }

    start_v2 = timerStart(start_v2);

    for (int k_count = 0; k_count < k; k_count++) {
        // 1st column
        sign_v2[0] = 0;
        sign_v2[(n + 2) * (n + 1)] = 0;
        for (int l = 0, i = n + 2, j = n + 2 + n; l < n; l++) {
            sign_v2[i] = sign_v2[j];
            i += n + 2;
            j += n + 2;
        }

        // 1st row
        for (int l = 0, i = 1, j = n * (n + 2) + 1; l < n; l++) {
            sign_v2[i] = sign_v2[j];
            i++;
            j++;
        }

        // 2nd column
        sign_v2[n + 1] = 0;
        sign_v2[(n + 2) * (n + 2) - 1] = 0;
        for (int l = 0, i = n + 2 + n + 1, j = n + 2 + 1; l < n; l++) {
            sign_v2[i] = sign_v2[j];
            i += n + 2;
            j += n + 2;
        }

        // 2nd row
        for (int l = 0, i = (n + 2) * (n + 1) + 1, j = (n + 2) + 1; l < n; l++) {
            sign_v2[i] = sign_v2[j];
            i++;
            j++;
        }

        // print sign_v2 array

        // for (int i = 0; i < sign_size; i++) {
        //     printf("%d\t", sign_v2[i]);
        //     if ((i + 1) % (n + 2) == 0) {
        //         printf("\n");
        //     }
        // }
        // printf("\n\n");

        // CUDA

        gpuErrchk(hipMalloc((void**)&ising_sign_d_v2, (n + 2) * (n + 2) * sizeof(int)));
        gpuErrchk(hipMemcpy(ising_sign_d_v2, sign_v2, (n + 2) * (n + 2) * sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMalloc((void**)&ising_out_v2, n * n * sizeof(int)));

        if ((nb_v2 * bs_v2 * b_v2 * b_v2) != (n * n)) {
            printf("error\n");
        }

        v2_kernel<<<nb_v2, bs_v2>>>(ising_sign_d_v2, ising_out_v2);
        ising_out_h_v2 = (int*)malloc(ising_sign_size * sizeof(int));
        gpuErrchk(hipMemcpy(ising_out_h_v2, ising_out_v2, ising_sign_size * sizeof(int), hipMemcpyDeviceToHost));

        // Print ising kernel output
        // printf("\t");
        // for (int i = 0; i < (ising_sign_size); i++) {
        //     printf("%d\t", ising_out_h_v2[i]);
        //     if ((i % n) == n - 1) {
        //         printf("\n\t");
        //     }
        // }
        // printf("\n\n");
        for (int i = 0, j = n + 3; i < ising_sign_size;) {
            for (int t_i = 0; t_i < n; t_i++, i++, j++) {
                sign_v2[j] = ising_out_h_v2[i];
            }
            j += 2;
        }

        // Free cuda memory
        hipFree(ising_sign_d_v2);
        hipFree(ising_out_v2);
    }
    stop_v2 = timerStop(stop_v2);

    free(sign_v2);

    printf("Time for V2: %lf seconds\n", timeDif(start_v2, stop_v2));

    printf("\nv3 starts\n\n");

    /*------------------------V3------------------------*/

    int* sign_v3;
    int* ising_out_h_v3;

    // CUDA lines
    int* ising_sign_d_v3;
    int* ising_out_v3;

    sign_v3 = (int*)malloc(sign_size * sizeof(int));

    for (int i = 0; i < sign_size; i++) {
        sign_v3[i] = sign[i];
    }

    start_v3 = timerStart(start_v3);

    for (int k_count = 0; k_count < k; k_count++) {
        // 1st column
        sign_v3[0] = 0;
        sign_v3[(n + 2) * (n + 1)] = 0;
        for (int l = 0, i = n + 2, j = n + 2 + n; l < n; l++) {
            sign_v3[i] = sign_v3[j];
            i += n + 2;
            j += n + 2;
        }

        // 1st row
        for (int l = 0, i = 1, j = n * (n + 2) + 1; l < n; l++) {
            sign_v3[i] = sign_v3[j];
            i++;
            j++;
        }

        // 2nd column
        sign_v3[n + 1] = 0;
        sign_v3[(n + 2) * (n + 2) - 1] = 0;
        for (int l = 0, i = n + 2 + n + 1, j = n + 2 + 1; l < n; l++) {
            sign_v3[i] = sign_v3[j];
            i += n + 2;
            j += n + 2;
        }

        // 2nd row
        for (int l = 0, i = (n + 2) * (n + 1) + 1, j = (n + 2) + 1; l < n; l++) {
            sign_v3[i] = sign_v3[j];
            i++;
            j++;
        }

        // print sign_v3 array

        // for (int i = 0; i < sign_size; i++) {
        //     printf("%d\t", sign_v3[i]);
        //     if ((i + 1) % (n + 2) == 0) {
        //         printf("\n");
        //     }
        // }
        // printf("\n\n");

        // CUDA

        gpuErrchk(hipMalloc((void**)&ising_sign_d_v3, sign_size * sizeof(int)));
        gpuErrchk(hipMemcpy(ising_sign_d_v3, sign_v3, sign_size * sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMalloc((void**)&ising_out_v3, sign_size * sizeof(int)));

        int block_elems = (sqrt((n * n) / nb_v3) + 2) * (sqrt((n * n) / nb_v3) + 2);
        int side_blocks = sqrt(nb_v3);
        int side_block_elems = sqrt(block_elems) - 2;

        if ((nb_v3 * bs_v3 * b_v3 * b_v3) != (n * n) && ((side_block_elems * side_block_elems) / bs_v3) == (b_v3 * b_v3)) {
            printf("error in assigned values\n");
        }

        v3_kernel<<<nb_v3, bs_v3>>>(ising_sign_d_v3, ising_out_v3, block_elems, side_blocks, side_block_elems);

        ising_out_h_v3 = (int*)malloc(sign_size * sizeof(int));
        gpuErrchk(hipMemcpy(ising_out_h_v3, ising_out_v3, sign_size * sizeof(int), hipMemcpyDeviceToHost));

        // Print ising kernel output

        // printf("\t");
        // for (int i = 0; i < sign_size; i++) {
        //     printf("%d\t", ising_out_h_v3[i]);
        //     if (((i + 1) % (n + 2)) == 0) {
        //         printf("\n\t");
        //     }
        // }
        // printf("\n\n");

        for (int i = 0; i < sign_size; i++) {
            sign_v3[i] = ising_out_h_v3[i];
        }

        // Free cuda memory
        hipFree(ising_sign_d_v3);
        hipFree(ising_out_v3);
        free(ising_out_h_v3);
    }

    stop_v3 = timerStop(stop_v3);

    free(sign_v3);
    printf("Time for V3: %lf seconds\n", timeDif(start_v3, stop_v3));
    return 0;
}

__global__ void v1_kernel(int* ising_sign_d_v1, int* ising_out_v1)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int line = (idx / n) + 1;
    int pos = (idx % n) + 1;

    // ising_out_v1 = self+left+right+up+down / abs(self+left+right+up+down)
    ising_out_v1[idx] = ising_sign_d_v1[line * (n + 2) + pos] + ising_sign_d_v1[line * (n + 2) + (pos - 1)] + ising_sign_d_v1[line * (n + 2) + (pos + 1)] + ising_sign_d_v1[(line - 1) * (n + 2) + pos] + ising_sign_d_v1[(line + 1) * (n + 2) + pos];
    ising_out_v1[idx] /= abs(ising_out_v1[idx]);
}

__global__ void v2_kernel(int* ising_sign_d_v2, int* ising_out_v2)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int blocks = b_v2 * b_v2;
    int k_id = idx * blocks;
    int line = 0;
    int pos = 0;
    for (int i = 0; i < blocks; i++) {
        line = ((k_id + i) / n) + 1;
        pos = ((k_id + i) % n) + 1;
        // self+left+right+up+down
        ising_out_v2[k_id + i] = ising_sign_d_v2[line * (n + 2) + pos] + ising_sign_d_v2[line * (n + 2) + (pos - 1)] + ising_sign_d_v2[line * (n + 2) + (pos + 1)] + ising_sign_d_v2[(line - 1) * (n + 2) + pos] + ising_sign_d_v2[(line + 1) * (n + 2) + pos];
        ising_out_v2[k_id + i] /= abs(ising_out_v2[k_id + i]);
    }
}

__global__ void v3_kernel(int* ising_sign_d_v3, int* ising_out_v3, int block_elems, int side_blocks, int side_block_elems)
{
    __shared__ int shared_mem[shared_size];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // example of 4 blocks on 8 * 8 = 64 array
    // threads inside cooperate to fill shared memory
    int thread_shared_count = block_elems / bs_v3;
    int idx_pos = (idx % bs_v3) * thread_shared_count; // 0-9-18-27
    int thread_shared_count_temp = thread_shared_count;
    if ((idx + 1) % bs_v3 == 0) {
        thread_shared_count += block_elems % thread_shared_count_temp;
    }
    // fill shared with block's values

    // each block:
    // 16 values
    // 4 threads
    // 2 * 2 bb
    // side_blocks = 2 (sqrt(nb))

    // so 4 blocks -> 2 lines 2 columns
    // block1 -> r(0-3)c(0-3)       0       0-3     -> 0-3 || 0-3   ->  0 || 0
    // block2 -> r(0-3)c(4-7)       4       4-7     -> 0-3 || 4-7   ->  0 || 1
    // block3 -> r(4-7)c(0-3)       8       8-11    -> 4-7 || 0-3   ->  1 || 0
    // block4 -> r(4-7)c(4-7)       12      12-15   -> 4-7 || 4-7   ->  1 || 1

    // Block's grid
    int line0 = bs_v3 * side_blocks; // 8
    int line = idx / line0;
    int column0 = idx % line0;
    int column = column0 / bs_v3;

    int line_index = ((side_block_elems * side_block_elems * side_blocks) + (side_block_elems * 2)) * line + n + 2 + 1;
    int column_index = column * (side_block_elems); // 0*4, 1*4

    int j = line_index + column_index - n - 2 - 1;

    int j_prev = 0;

    if (idx_pos > 0) {
        j_prev = idx_pos / (side_block_elems + 2) * (n + 2) + idx_pos % (side_block_elems + 2);
    }
    j += j_prev;

    // Fill shared_mem

    for (int i = idx_pos; i < idx_pos + thread_shared_count; i++, j++) {
        shared_mem[i] = ising_sign_d_v3[j];

        if ((i + 1) % (side_block_elems + 2) == 0) {
            j = j + n + 2 - side_block_elems - 2;
        }
    }
    __syncthreads();

    // Find ising model in shared

    j = line_index + column_index - n - 2 - 1;

    j_prev = 0;

    if (idx_pos > 0) {
        j_prev = idx_pos / (side_block_elems + 2) * (n + 2) + idx_pos % (side_block_elems + 2);
    }
    j += j_prev;

    for (int i = idx_pos; i < idx_pos + thread_shared_count; i++, j++) {
        if ((i > side_block_elems + 2) && (i < (block_elems - side_block_elems - 2 - 1)) && ((i + 1) % (side_block_elems + 2) != 0) && (i % (side_block_elems + 2) != 0)) {
            ising_out_v3[j] = shared_mem[i] + shared_mem[i - 1] + shared_mem[i + 1] + shared_mem[i - side_block_elems - 2] + shared_mem[i + side_block_elems + 2];
            ising_out_v3[j] /= abs(ising_out_v3[j]);
        } else {
            ising_out_v3[j] = shared_mem[i];
        }
        if ((i + 1) % (side_block_elems + 2) == 0) {
            j = j + n + 2 - side_block_elems - 2;
        }
    }
    __syncthreads();
}

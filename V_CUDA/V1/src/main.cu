#include "hip/hip_runtime.h"
% % cu
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define n 8
#define k 2

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

    inline void
    gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void vector_add(int* d_out, int* ising_sign_d, size_t pitch, int j);

int main(int argc, char* argv[])
{
    int** sign;
    int** ising_sign;
    int** ising_sign_h;
    // CUDA lines
    int* ising_sign_d;
    size_t pitch;
    time_t t;

    /* Intializes random number generator */
    srand((unsigned)time(&t));

    // Malloc 2D Arrays

    sign = (int**)malloc((n + 2) * sizeof(int*));

    for (int i = 0; i < n + 2; i++) {
        sign[i] = (int*)malloc((n + 2) * sizeof(int));
    }

    ising_sign = (int**)malloc(n * sizeof(int*));

    for (int i = 0; i < n; i++) {
        ising_sign[i] = (int*)malloc(n * sizeof(int));
    }

    ising_sign_h = (int**)malloc(3 * sizeof(int*));

    for (int i = 0; i < 3; i++) {
        ising_sign_h[i] = (int*)malloc((n + 2) * sizeof(int));
    }

    // Could use module but better surround the array with 1 line of values
    // Example cost of 40000 X 40000 array in CPI

    // Initialize 2D array

    for (int i = 1; i < n + 1; i++) {
        for (int j = 1; j < n + 1; j++) {
            sign[i][j] = 1 - (2 * (rand() % 2));
        }
        printf("\n");
    }

    for (int count = 0; count < k; count++) {
        // boundaries set

        // 1st column
        sign[0][0] = 0;
        sign[n + 1][0] = 0;
        for (int i = 0; i < n; i++) {
            sign[i + 1][0] = sign[i + 1][n];
        }

        // 1st row
        sign[0][n + 1] = 0;
        for (int i = 0; i < n; i++) {
            sign[0][i + 1] = sign[n][i + 1];
        }

        // 2nd column
        sign[n + 1][n + 1] = 0;
        for (int i = 0; i < n; i++) {
            sign[i + 1][n + 1] = sign[i + 1][1];
        }

        // 2nd row
        for (int i = 0; i < n; i++) {
            sign[n + 1][i + 1] = sign[1][i + 1];
        }

        // print sign array
        for (int i = 0; i < n + 2; i++) {
            for (int j = 0; j < n + 2; j++) {
                printf("%d\t", sign[i][j]);
            }
            printf("\n");
        }
        printf("\n\n");

        // CUDA

        gpuErrchk(hipMallocPitch(&ising_sign_d, &pitch, (n + 2) * sizeof(int), 3));

        int* d_out;
        gpuErrchk(hipMalloc((void**)&d_out, sizeof(int)));

        int h_out;

        for (int i = 0; i < n; i++) {
            printf("\t");
            // fill 2d ising_sign_h array with 3 lines
            for (int r = 0; r < 3; r++) {
                for (int l = 0; l < n + 2; l++) {
                    ising_sign_h[r][l] = sign[i + r][l];
                }
            }
            // TODO: check hipMemcpy2D
            //gpuErrchk(hipMemcpy2D(ising_sign_d, pitch, ising_sign_h, n * sizeof(int), n * sizeof(int), 3, hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy2D(ising_sign_d, pitch, ising_sign_h, (n + 2) * sizeof(int), (n + 2) * sizeof(int), 3, hipMemcpyHostToDevice));

            for (int j = 0; j < (n); j++) {
                ising_sign[i][j] = sign[i + 1][j + 1] + sign[i - 1 + 1][j + 1] + sign[i + 1][j - 1 + 1] + sign[i + 1 + 1][j + 1] + sign[i + 1][j + 1 + 1];
                ising_sign[i][j] /= abs(ising_sign[i][j]);

                vector_add<<<1, 1>>>(d_out, ising_sign_d, pitch, j);
                gpuErrchk(hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost));

                printf("cd:%d|", h_out);
                printf("%d\t", ising_sign[i][j]);
            }
            printf("\n");
        }
        printf("\n");
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                sign[i + 1][j + 1] = ising_sign[i][j];
            }
        }
    }

    // free memory

    for (int i = 0; i < n + 2; i++) {
        free(sign[i]);
    }
    free(sign);

    for (int i = 0; i < n; i++) {
        free(ising_sign[i]);
    }
    free(ising_sign);

    return 0;
}

__global__ void vector_add(int* d_out, int* ising_sign_d, size_t pitch, int j)
{
    //d_out = ising_sign_d[1][j + 1] + ising_sign_d[0][j + 1] + ising_sign_d[1][j + 2] + ising_sign_d[2][j + 1] + ising_sign_d[1][j];

    *d_out = 0;
    int* row0 = (int*)((char*)ising_sign_d + 0 * pitch);
    printf("heyhey: %d|%d\n", j, row0[j + 1]);
    //*d_out = row0[j + 1];
    // int* row1 = (int*)((char*)ising_sign_d + 1 * pitch);
    // *d_out += row1[j];
    // *d_out += row1[j + 1];
    // *d_out += row1[j + 2];
    // int* row2 = (int*)((char*)ising_sign_d + 2 * pitch);
    // *d_out += row2[j + 1];

    // *d_out /= abs(*d_out);
    //*d_out = 5;
}

//! pip install git+git://github.com/andreinechaev/nvcc4jupyter.git
//%load_ext nvcc_plugin
//% % cu
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define n 6
#define k 5

#define nb 12
#define bs 3

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

__global__ void vector_add(int* ising_sign_d, int* ising_out, int N);

int main(int argc, char* argv[])
{
    int* sign;
    int* ising_out_h;

    // CUDA lines
    int* ising_sign_d;
    int* ising_out;

    time_t t;

    /* Intializes random number generator */
    srand((unsigned)time(&t));

    int sign_size = (n + 2) * (n + 2);
    int ising_sign_size = n * n;
    // Malloc 1D Arrays

    sign = (int*)malloc(sign_size * sizeof(int));

    // Could use module but better surround the array with 1 line of values
    // Example cost of 40000 X 40000 array in CPI

    // Initialize 2D array

    for (int i = 0; i < sign_size; i++) {
        sign[i] = 1 - (2 * (rand() % 2));
    }
    printf("\n");

    // boundaries set

    for (int k_count = 0; k_count < k; k_count++) {
        printf("\n\n\nIteration_%d:\n\n", k_count + 1);
        // 1st column
        sign[0] = 0;
        sign[(n + 2) * (n + 1)] = 0;
        for (int l = 0, i = n + 2, j = n + 2 + n; l < n; l++) {
            sign[i] = sign[j];
            i += n + 2;
            j += n + 2;
        }

        // 1st row
        for (int l = 0, i = 1, j = n * (n + 2) + 1; l < n; l++) {
            sign[i] = sign[j];
            i++;
            j++;
        }

        // 2nd column
        sign[n + 1] = 0;
        sign[(n + 2) * (n + 2) - 1] = 0;
        for (int l = 0, i = n + 2 + n + 1, j = n + 2 + 1; l < n; l++) {
            sign[i] = sign[j];
            i += n + 2;
            j += n + 2;
        }

        // 2nd row
        for (int l = 0, i = (n + 2) * (n + 1) + 1, j = (n + 2) + 1; l < n; l++) {
            sign[i] = sign[j];
            i++;
            j++;
        }

        // print sign array
        for (int i = 0; i < sign_size; i++) {
            printf("%d\t", sign[i]);
            if ((i + 1) % (n + 2) == 0) {
                printf("\n");
            }
        }
        printf("\n\n");

        // CUDA

        gpuErrchk(hipMalloc((void**)&ising_sign_d, (n + 2) * (n + 2) * sizeof(int)));
        gpuErrchk(hipMemcpy(ising_sign_d, sign, (n + 2) * (n + 2) * sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMalloc((void**)&ising_out, n * n * sizeof(int)));

        vector_add<<<nb, bs>>>(ising_sign_d, ising_out, n);
        ising_out_h = (int*)malloc(ising_sign_size * sizeof(int));
        gpuErrchk(hipMemcpy(ising_out_h, ising_out, ising_sign_size * sizeof(int), hipMemcpyDeviceToHost));

        // Print ising kernel output
        printf("\t");
        for (int i = 0; i < (nb * bs); i++) {
            printf("%d\t", ising_out_h[i]);
            if ((i % n) == n - 1) {
                printf("\n\t");
            }
        }
        printf("\n\n");
        for (int i = 0, j = n + 3; i < ising_sign_size;) {
            for (int t_i = 0; t_i < n; t_i++, i++, j++) {
                sign[j] = ising_out_h[i];
            }
            j += 2;
        }

        // Free cuda memory
        hipFree(ising_sign_d);
        hipFree(ising_out);
    }
    free(sign);
    return 0;
}

__global__ void vector_add(int* ising_sign_d, int* ising_out, int N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int line = (idx / N) + 1;
    int pos = (idx % N) + 1;

    // ising_out = self+left+right+up+down / abs(self+left+right+up+down)
    ising_out[idx] = ising_sign_d[line * (N + 2) + pos] + ising_sign_d[line * (N + 2) + (pos - 1)] + ising_sign_d[line * (N + 2) + (pos + 1)] + ising_sign_d[(line - 1) * (N + 2) + pos] + ising_sign_d[(line + 1) * (N + 2) + pos];
    ising_out[idx] /= abs(ising_out[idx]);
}
